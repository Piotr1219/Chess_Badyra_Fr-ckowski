#include "hip/hip_runtime.h"
#include "EvaluateMovesCuda.cuh"
#include <assert.h>
//#include <stdio.h>
#include <chrono>


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


__global__ void evaluate_moves_kernel(Square* squares, int* each_square_possible_moves, Position* moves) {
	int block_index = blockIdx.x;
	int cell_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (squares[block_index].Piece1.PieceType != ChessPieceType::None) {
		if (each_square_possible_moves[cell_index] > 0 && each_square_possible_moves[cell_index] < 64) {
			//moves[cell_index] = Position();

			moves[cell_index].SrcPosition = block_index;
			moves[cell_index].DstPosition = each_square_possible_moves[cell_index];

			//Piece pieceAttacked = squares[moves[cell_index].DstPosition].Piece1;
			//moves[cell_index].Score += pieceAttacked.PieceValue - squares[block_index].Piece1.PieceValue;

			// move.Score += squares[move.DstPosition].Piece1.Moved
			//moves[cell_index].Score += squares[block_index].Piece1.PieceActionValue;
			//printf("Move from %d to %d\n", block_index, each_square_possible_moves[cell_index]);
		}
		else {
			moves[cell_index].SrcPosition = -1;
		}
	}
	else {
		moves[cell_index].SrcPosition = -1;
	}
}


namespace EvaluateMoves {
	list<Position> EvaluateMovesCuda(Board& examineBoard, char depth) {
		// first we need to copy all moves from each piece lists to array
		std::chrono::steady_clock::time_point begin1 = std::chrono::steady_clock::now();
		int* each_squares_possible_moves = new int[64 * 32];

		for (int i = 0; i < 64; ++i) {
			Piece piece = examineBoard.Squares[i].Piece1;
			if (piece.PieceType == ChessPieceType::None)
				continue;

			//Make sure the color is the same color as the one we are moving.
			if (piece.PieceColor != examineBoard.WhoseMove)
				continue;
			std::copy(piece.ValidMoves.begin(), piece.ValidMoves.end(), each_squares_possible_moves + 32 * i);
		}
		std::chrono::steady_clock::time_point end1 = std::chrono::steady_clock::now();
		std::cout << "Time for list to array conversion: " << std::chrono::duration_cast<std::chrono::microseconds>(end1 - begin1).count() << "[�s]" << std::endl;

		std::chrono::steady_clock::time_point begin2 = std::chrono::steady_clock::now();
		Square* d_squares;
		int* d_each_square_possible_moves;
		Position* d_moves;
		Position* moves = (Position*)malloc(64 * 32 * sizeof(Position));

		hipMalloc(&d_squares, 64 * sizeof(Square));
		hipMalloc(&d_each_square_possible_moves, 64 * 32 * sizeof(int));
		hipMalloc(&d_moves, 64 * 32 * sizeof(Position));

		hipMemcpy(d_squares, examineBoard.Squares, 64 * sizeof(Square), hipMemcpyHostToDevice);
		hipMemcpy(d_each_square_possible_moves, each_squares_possible_moves, 64 * 32 * sizeof(int), hipMemcpyHostToDevice);
		std::chrono::steady_clock::time_point end2 = std::chrono::steady_clock::now();
		std::cout << "Time for data upload: " << std::chrono::duration_cast<std::chrono::microseconds>(end2 - begin2).count() << "[�s]" << std::endl;

		std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
		evaluate_moves_kernel << <64, 32 >> > (d_squares, d_each_square_possible_moves, d_moves);
		std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
		std::cout << "Time for cuda kernel only: " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[�s]" << std::endl;
		hipMemcpy(moves, d_moves, 64 * 32 * sizeof(Position), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		hipFree(d_squares), hipFree(d_each_square_possible_moves), hipFree(d_moves);
		
		list<Position> positions = list<Position>();
		for (int i = 0; i < 64 * 32; ++i) {
			if (moves[i].SrcPosition != -1) {
				positions.push_back(moves[i]);
			}
		}

		return positions;
	}
}
