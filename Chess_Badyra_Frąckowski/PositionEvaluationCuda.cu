#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include "PositionEvaluationCuda.cuh"
#include <assert.h>
//#include <stdio.h>
#include <chrono>
#ifndef __HIPCC__  
	#define __HIPCC__
#endif

extern long time_cuda = 0;
extern long count_positions = 0;
extern long min_time = 1000000000;
extern long max_time = 0;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void sum_matrix(int* result) {
	__shared__ int score[64];

	int index = threadIdx.x;
	score[index] = result[index];

	__syncthreads();
	{
		//printf("Score from index %d is %d\n", index, score[index]);
		for (int stride = 32; stride > 0; stride >>= 1) {
			//__syncthreads();
			if (index < stride) {
				score[index] = score[index] + score[index + stride];
				//printf("Score from index %d is %d\n", index, score[index]);
			}
			__syncthreads();
		}

		result[0] = score[0];
	}
}

// square is a whole board
// 64 threads
// position is same thing as index, wont be needed
// endGamePhase is also same for all positions, neverhteless not sure if use it here, many ifs
// insufficientMaterial wyrzucamy, bardzo du�o wyst�puje, lepiej zawsze dla roota to sprawdza�
// many things including pawns, but leaving it unchanged for now
// also bishop count and knigh count wont work, cannot iterate over board for it!!!
// same thing for pawns count

__device__ static short PawnTableGPU[64] =
{
		0,  0,  0,  0,  0,  0,  0,  0,
		50, 50, 50, 50, 50, 50, 50, 50,
		20, 20, 30, 40, 40, 30, 20, 20,
		5,  5, 10, 30, 30, 10,  5,  5,
		0,  0,  0, 25, 25,  0,  0,  0,
		5, -5,-10,  0,  0,-10, -5,  5,
		5, 10, 10,-30,-30, 10, 10,  5,
		0,  0,  0,  0,  0,  0,  0,  0
};
__device__ static short KnightTableGPU[64] =
{
	-50,-40,-30,-30,-30,-30,-40,-50,
	-40,-20,  0,  0,  0,  0,-20,-40,
	-30,  0, 10, 15, 15, 10,  0,-30,
	-30,  5, 15, 20, 20, 15,  5,-30,
	-30,  0, 15, 20, 20, 15,  0,-30,
	-30,  5, 10, 15, 15, 10,  5,-30,
	-40,-20,  0,  5,  5,  0,-20,-40,
	-50,-30,-20,-30,-30,-20,-30,-50,
};

__device__ static short BishopTableGPU[64] =
{
	-20,-10,-10,-10,-10,-10,-10,-20,
	-10,  0,  0,  0,  0,  0,  0,-10,
	-10,  0,  5, 10, 10,  5,  0,-10,
	-10,  5,  5, 10, 10,  5,  5,-10,
	-10,  0, 10, 10, 10, 10,  0,-10,
	-10, 10, 10, 10, 10, 10, 10,-10,
	-10,  5,  0,  0,  0,  0,  5,-10,
	-20,-10,-40,-10,-10,-40,-10,-20,
};

__device__ static short KingTableGPU[64] =
{
	-30, -40, -40, -50, -50, -40, -40, -30,
	-30, -40, -40, -50, -50, -40, -40, -30,
	-30, -40, -40, -50, -50, -40, -40, -30,
	-30, -40, -40, -50, -50, -40, -40, -30,
	-20, -30, -30, -40, -40, -30, -30, -20,
	-10, -20, -20, -20, -20, -20, -20, -10,
	20,  20,   0,   0,   0,   0,  20,  20,
	20,  30,  10,   0,   0,  10,  30,  20
};

__device__ static short KingTableEndGameGPU[64] =
{
	-50,-40,-30,-20,-20,-30,-40,-50,
	-30,-20,-10,  0,  0,-10,-20,-30,
	-30,-10, 20, 30, 30, 20,-10,-30,
	-30,-10, 30, 40, 40, 30,-10,-30,
	-30,-10, 30, 40, 40, 30,-10,-30,
	-30,-10, 20, 30, 30, 20,-10,-30,
	-30,-30,  0,  0,  0,  0,-30,-30,
	-50,-30,-30,-30,-30,-30,-30,-50
};
/*
__host__ __device__ static short Kings[64];
__host__ __device__ static short Queens[64];
__host__ __device__ static short Rooks[64];
__host__ __device__ static short Bishops[64];
__host__ __device__ static short Knights[64];
__host__ __device__ static short Pawns[64];
*/


__global__ void evaluate_board_kernel(Square* squares, bool endGamePhase, int* result) 
{	
	/*
	__shared__ short PawnTable[64];
	__shared__ short KnightTable[64];
	__shared__ short BishopTable[64];
	__shared__ short KingTable[64];
	__shared__ short KingTableEndGame[64];
	*/
		__shared__ int score[64];

		int index = threadIdx.x;
		int position = index;
		/*
		PawnTable[index] = PawnTableGPU[index];
		KnightTable[index] = KnightTableGPU[index];
		BishopTable[index] = BishopTableGPU[index];
		KingTable[index] = KingTableGPU[index];
		KingTableEndGame[index] = KingTableEndGameGPU[index];
		*/
	{
		if (squares[index].Piece1.PieceColor == ChessPieceColor::Black)
		{
			position = (63 - index);
		}

		if (squares[index].Piece1.PieceType == ChessPieceType::None)
		{
			score[index] = 0;
			//printf("Assigned to index %d value %d\n", index, score[index]);
		}
		else {

			//Calculate Piece Values
			score[index] += squares[index].Piece1.PieceValue;
			//printf("first assignment: %d\n", score[index]);
			score[index] += squares[index].Piece1.DefendedValue;
			//printf("second assignment: %d\n", score[index]);
			score[index] -= squares[index].Piece1.AttackedValue;
			//printf("third assignment: %d\n", score[index]);

			//Double Penalty for Hanging Pieces
			if (squares[index].Piece1.DefendedValue < squares[index].Piece1.AttackedValue)
			{
				score[index] -= ((squares[index].Piece1.AttackedValue - squares[index].Piece1.DefendedValue) * 10);
				//printf("forth assignment: %d\n", score[index]);
			}

			//Add Points for Mobility
			/*
			if (!squares[index].Piece1.ValidMoves.empty())
			{
				score[index] += squares[index].Piece1.ValidMoves.size();
			}
			*/

			// LastValidMoveCount is not always initialized!!!! So it gives error somethimes
			//score[index] += squares[index].Piece1.LastValidMoveCount;
			//printf("fifth assignment: %d\n", score[index]);

			if (squares[index].Piece1.PieceType == ChessPieceType::Pawn)
			{

				if (index % 8 == 0 || index % 8 == 7)
				{
					//Rook Pawns are worth 15% less because they can only attack one way
					score[index] -= 15;
					//printf("sixht assignment: %d\n", score[index]);
				}

				//Calculate Position Values
				score[index] += PawnTableGPU[position];
				//printf("seventh assignment: %d\n", score[index]);
			}
			else if (squares[index].Piece1.PieceType == ChessPieceType::Knight)
			{
				//*knightCount = (knightCount);

				score[index] += KnightTableGPU[position];

				//In the end game remove a few points for Knights since they are worth less
				if (endGamePhase)
				{
					score[index] -= 10;
				}

			}
			else if (squares[index].Piece1.PieceType == ChessPieceType::Bishop)
			{
				//In the end game Bishops are worth more
				if (endGamePhase)
				{
					score[index] += 10;
				}

				score[index] += BishopTableGPU[position];
			}
			else if (squares[index].Piece1.PieceType == ChessPieceType::Queen)
			{
				if (squares[index].Piece1.Moved && !endGamePhase)
				{
					score[index] -= 10;
				}
			}
			else if (squares[index].Piece1.PieceType == ChessPieceType::King)
			{
				/*
				if (!squares[index].Piece1.ValidMoves.empty())
				{
					if (squares[index].Piece1.ValidMoves.size() < 2)
					{
						score[index] -= 5;
					}
				}
				*/
				if (squares[index].Piece1.LastValidMoveCount < 2)
				{
					score[index] -= 5;
				}

				if (endGamePhase)
				{
					score[index] += KingTableEndGameGPU[position];
				}
				else
				{
					score[index] += KingTableGPU[position];
				}
			}
			if (squares[index].Piece1.PieceColor == ChessPieceColor::Black) {
				score[index] = -score[index];
			}
		}
		//printf("Ended node: %d\n", index);
	}

	//__shared__ int results[64];
	//results[index] = score[index];
	__syncthreads();
	/*
	{
		//printf("Score from index %d is %d\n", index, score[index]);
		for (int stride = 32; stride > 0; stride >>= 1) {
			//__syncthreads();
			if (index < stride) {
				score[index] = score[index] + score[index + stride];
				//printf("Score from index %d is %d\n", index, score[index]);
			}
			__syncthreads();
		}

		result[0] = score[0];

	}
	*/
	
	sum_matrix << <1, 64 >> >(score);

}

__global__ void evaluate_board_kernel_v2(int endGamePhase, int* score, short* Kings, short* Queens, short* Rooks, short* Bishops, short* Knights, short* Pawns)
{
	__shared__ short PawnScore[64];
	__shared__ short KnightScore[64];
	__shared__ short BishopScore[64];
	__shared__ short RookScore[64];
	__shared__ short QueenScore[64];
	__shared__ short KingScore[64];
	__shared__ short KingScoreEndGame[64];

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int position_w = index;
	int position_b = 63 - index;

	
	PawnScore[index] = Pawns[position_w];
	PawnScore[index] -= Pawns[position_b];
	PawnScore[index] = PawnScore[index] * Pawns[index];

	KnightScore[index] = Knights[position_w];
	KnightScore[index] -= Knights[position_b];
	KnightScore[index] = KnightScore[index] * Knights[index];

	BishopScore[index] = Bishops[position_w];
	BishopScore[index] -= Bishops[position_b];
	BishopScore[index] = BishopScore[index] * Bishops[index];

	RookScore[index] = Rooks[index];

	QueenScore[index] = Queens[index];

	KingScore[index] = Kings[position_w];
	KingScore[index] -= Kings[position_b];
	KingScore[index] = KingScore[index] * Kings[index];
	/*
	KingScoreEndGame[index] = Kings[position_w];
	KingScoreEndGame[index] -= Kings[position_b];
	KingScoreEndGame[index] = KingScoreEndGame[index] * Kings[index] * endGamePhase;


	score[index] = PawnScore[index] + KnightScore[index] + BishopScore[index] + RookScore[index] + QueenScore[index] + KingScore[index] + KingScoreEndGame[index];


	/*
	if (squares[index].Piece1.PieceType == ChessPieceType::None)
	{
		score[index] = 0;
	}
	else {

		//Calculate Piece Values
		score[index] += squares[index].Piece1.PieceValue;
		//printf("first assignment: %d\n", score[index]);
		score[index] += squares[index].Piece1.DefendedValue;
		//printf("second assignment: %d\n", score[index]);
		score[index] -= squares[index].Piece1.AttackedValue;
		//printf("third assignment: %d\n", score[index]);

		//Double Penalty for Hanging Pieces
		if (squares[index].Piece1.DefendedValue < squares[index].Piece1.AttackedValue)
		{
			score[index] -= ((squares[index].Piece1.AttackedValue - squares[index].Piece1.DefendedValue) * 10);
			//printf("forth assignment: %d\n", score[index]);
		}

		//Add Points for Mobility
		/*
		if (!squares[index].Piece1.ValidMoves.empty())
		{
			score[index] += squares[index].Piece1.ValidMoves.size();
		}
		*/

		// LastValidMoveCount is not always initialized!!!! So it gives error somethimes
		//score[index] += squares[index].Piece1.LastValidMoveCount;
		//printf("fifth assignment: %d\n", score[index]);
		/*
		if (squares[index].Piece1.PieceType == ChessPieceType::Pawn)
		{

			if (index % 8 == 0 || index % 8 == 7)
			{
				//Rook Pawns are worth 15% less because they can only attack one way
				score[index] -= 15;
				//printf("sixht assignment: %d\n", score[index]);
			}

			//Calculate Position Values
			score[index] += PawnTable[position];
			//printf("seventh assignment: %d\n", score[index]);
		}
		else if (squares[index].Piece1.PieceType == ChessPieceType::Knight)
		{
			//*knightCount = (knightCount);

			score[index] += KnightTable[position];

			//In the end game remove a few points for Knights since they are worth less
			if (endGamePhase)
			{
				score[index] -= 10;
			}

		}
		else if (squares[index].Piece1.PieceType == ChessPieceType::Bishop)
		{
			//In the end game Bishops are worth more
			if (endGamePhase)
			{
				score[index] += 10;
			}

			score[index] += BishopTable[position];
		}
		else if (squares[index].Piece1.PieceType == ChessPieceType::Queen)
		{
			if (squares[index].Piece1.Moved && !endGamePhase)
			{
				score[index] -= 10;
			}
		}
		else if (squares[index].Piece1.PieceType == ChessPieceType::King)
		{
			/*
			if (!squares[index].Piece1.ValidMoves.empty())
			{
				if (squares[index].Piece1.ValidMoves.size() < 2)
				{
					score[index] -= 5;
				}
			}
			*/
			/*
			if (squares[index].Piece1.LastValidMoveCount < 2)
			{
				score[index] -= 5;
			}

			if (endGamePhase)
			{
				score[index] += KingTableEndGame[position];
			}
			else
			{
				score[index] += KingTable[position];
			}
		}
		if (squares[index].Piece1.PieceColor == ChessPieceColor::Black) {
			score[index] = -score[index];
		}
	}
	*/
	//printf("Ended node: %d\n", index);
	

}

namespace EvaluatePieces {

	void EvaluatePiecesCuda(Board& board) {
		std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
		//Black Score - 
		//White Score +
		/*
		short Kings[64];
		short Queens[64];
		short Rooks[64];
		short Bishops[64];
		short Knights[64];
		short Pawns[64];

		for (int i = 0; i < 64; ++i) {
			Kings[i] = 0;
			Queens[i] = 0;
			Rooks[i] = 0;
			Bishops[i] = 0;
			Knights[i] = 0;
			Pawns[i] = 0;

			switch (board.Squares[i].Piece1.PieceType) {
			case ChessPieceType::King:
				Kings[i] = board.Squares[i].Piece1.PieceValue / abs(board.Squares[i].Piece1.PieceValue);
				break;
			case ChessPieceType::Queen:
				Queens[i] = board.Squares[i].Piece1.PieceValue / abs(board.Squares[i].Piece1.PieceValue);
				break;
			case ChessPieceType::Rook:
				Rooks[i] = board.Squares[i].Piece1.PieceValue / abs(board.Squares[i].Piece1.PieceValue);
				break;
			case ChessPieceType::Bishop:
				Bishops[i] = board.Squares[i].Piece1.PieceValue / abs(board.Squares[i].Piece1.PieceValue);
				break;
			case ChessPieceType::Knight:
				Knights[i] = board.Squares[i].Piece1.PieceValue / abs(board.Squares[i].Piece1.PieceValue);
				break;
			case ChessPieceType::Pawn:
				Pawns[i] = board.Squares[i].Piece1.PieceValue / abs(board.Squares[i].Piece1.PieceValue);
				break;
			}
		}
		*/

		board.Score = 0;
		board.InsufficientMaterial = false;             // temporary, until not fixed how to assume if it is true
		
		/*

		char blackBishopCount = 0;
		char whiteBishopCount = 0;

		char blackKnightCount = 0;
		char whiteKnightCount = 0;


		char knightCount = 0;
		*/

		//fill_n(blackPawnCount, 8, 0);
		//blackPawnCount = new short[8];
		//whitePawnCount = new short[8];

		// for now also not including pawns computations until we find a way that does not include iterating over whole board (or we may add it later on CPU only)

		/*

		short blackPawnCount[8];
		short whitePawnCount[8];
		for (int i = 0; i < 8; i++) {
			blackPawnCount[i] = 0;
			whitePawnCount[i] = 0;
		}

		*/

		// add cuda below

		// needed structures and objects:
		// Square class needs in fact Piece class, and two enum types ChessPieceColor, ChessPieceType
		// board score propably will be replaced as 64 element int pointer for score, and bool for EndGamePhase, after cuda we will assign each board its score
		// EvaluatePieceScore2 seems to have only regular data types
		// CheckPawnWall functions have to many if instructions, for now will not include them in evaluation results

		/*
		for (int x = 0; x < 64; x++)
		{
			Square square = board.Squares[x];

			if (square.Piece1.PieceType == ChessPieceType::None)
			{
				continue;       // TODO: is it really necessary here? Does nothing i guess
			}


			if (square.Piece1.PieceColor == ChessPieceColor::White)
			{
				board.Score += EvaluatePieceScore2(square, x, board.EndGamePhase,
					whiteKnightCount, whiteBishopCount, insufficientMaterial, blackPawnCount, whitePawnCount);

				if (square.Piece1.PieceType == ChessPieceType::King)
				{
					if (x != 59 && x != 60)
					{
						int pawnPos = x - 8;

						board.Score += CheckPawnWall2(board, pawnPos, x);

						pawnPos = x - 7;

						board.Score += CheckPawnWall2(board, pawnPos, x);

						pawnPos = x - 9;

						board.Score += CheckPawnWall2(board, pawnPos, x);
					}
				}
			}
			else if (square.Piece1.PieceColor == ChessPieceColor::Black)
			{
				board.Score -= EvaluatePieceScore2(square, x, board.EndGamePhase,
					blackKnightCount, blackBishopCount, insufficientMaterial, blackPawnCount, whitePawnCount);


				if (square.Piece1.PieceType == ChessPieceType::King)
				{
					if (x != 3 && x != 4)
					{
						int pawnPos = x + 8;

						board.Score -= CheckPawnWall2(board, pawnPos, x);

						pawnPos = x + 7;

						board.Score -= CheckPawnWall2(board, pawnPos, x);

						pawnPos = x + 9;

						board.Score -= CheckPawnWall2(board, pawnPos, x);
					}

				}

			}

			if (square.Piece1.PieceType == ChessPieceType::Knight)
			{
				knightCount = (knightCount + 1);

				if (knightCount > 1)
				{
					insufficientMaterial = false;
				}
			}

			if ((blackBishopCount + whiteBishopCount) > 1)
			{
				insufficientMaterial = false;
			}
			else if ((blackBishopCount + blackKnightCount) > 1)
			{
				insufficientMaterial = false;
			}
			else if ((whiteBishopCount + whiteKnightCount) > 1)
			{
				insufficientMaterial = false;
			}

		}

		// until that point
		*/

		// not including insufficientMaterial for now, gotta calculate it outside GPU
		/*
		if (insufficientMaterial)
		{
			board.Score = 0;
			board.StaleMate = true;
			board.InsufficientMaterial = true;
			return;
		}
		*/
		int* d_score;
		int* score = (int*)malloc(sizeof(int));
		/*
		short* d_Kings;
		short* d_Queens;
		short* d_Rooks;
		short* d_Bishops;
		short* d_Knights;
		short* d_Pawns;
		*/
		//short* d_PawnTable;
		Square* d_squares;
		

		hipMalloc(&d_score, sizeof(int));
		hipMalloc(&d_squares, sizeof(board.Squares));
		/*
		hipMalloc(&d_Kings, 64 * sizeof(short));
		hipMalloc(&d_Queens, 64 * sizeof(short));
		hipMalloc(&d_Rooks, 64 * sizeof(short));
		hipMalloc(&d_Bishops, 64 * sizeof(short));
		hipMalloc(&d_Knights, 64 * sizeof(short));
		hipMalloc(&d_Pawns, 64 * sizeof(short));
		*/

		hipMemcpy(d_squares, board.Squares, sizeof(board.Squares), hipMemcpyHostToDevice);
		/*
		hipMemcpy(d_Kings, Kings, 64 * sizeof(short), hipMemcpyHostToDevice);
		hipMemcpy(d_Queens, Queens, 64 * sizeof(short), hipMemcpyHostToDevice);
		hipMemcpy(d_Rooks, Rooks, 64 * sizeof(short), hipMemcpyHostToDevice);
		hipMemcpy(d_Bishops, Bishops, 64 * sizeof(short), hipMemcpyHostToDevice);
		hipMemcpy(d_Knights, Knights, 64 * sizeof(short), hipMemcpyHostToDevice);
		hipMemcpy(d_Pawns, Pawns, 64 * sizeof(short), hipMemcpyHostToDevice);
		*/

		/*
		int late_game = 0;
		if (board.EndGamePhase) {
			late_game = 1;
		}
		*/

		evaluate_board_kernel << <1, 64 >> > (d_squares, board.EndGamePhase, d_score);
		//evaluate_board_kernel_v2 << <1, 64 >> > (late_game, d_score, d_Kings, d_Queens, d_Rooks, d_Bishops, d_Knights, d_Pawns);
		hipMemcpy(score, d_score, sizeof(int), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		if (board.StaleMate)
		{
			return;
		}
		if (board.HalfMoveClock >= 100)
		{
			return;
		}
		if (board.RepeatedMove >= 3)
		{
			return;
		}
		if (board.BlackMate)
		{
			board.Score = 32767;
			return;
		}
		if (board.WhiteMate)
		{
			board.Score = -32767;
			return;
		}
		if (board.BlackCheck)
		{
			board.Score += 70;
			if (board.EndGamePhase)
				board.Score += 10;
		}
		else if (board.WhiteCheck)
		{
			board.Score -= 70;
			if (board.EndGamePhase)
				board.Score -= 10;
		}
		if (board.BlackCastled)
		{
			board.Score -= 50;
		}
		if (board.WhiteCastled)
		{
			board.Score += 50;
		}
		//Add a small bonus for tempo (turn)
		if (board.WhoseMove == ChessPieceColor::White)
		{
			board.Score += 10;
		}
		else
		{
			board.Score -= 10;
		}
		if (board.EndGamePhase)
		{
			if (board.BlackCheck)
			{
				board.Score += 10;
			}
			else if (board.WhiteCheck)
			{
				board.Score -= 10;
			}
		}
		else
		{
			if (!board.WhiteCanCastle && !board.WhiteCastled)
			{
				board.Score -= 50;
			}
			if (!board.BlackCanCastle && !board.BlackCastled)
			{
				board.Score += 50;
			}
		}
		//board.Score = *score;
		//for (int i = 0; i < 64; ++i) {
			//board.Score += score[i];
			//printf("Score from i: %d is: %d\n", i, score[i]);
		//}
		board.Score = *score;
		/*
		//Black Isolated Pawns
		if (blackPawnCount[0] >= 1 && blackPawnCount[1] == 0)
		{
			board.Score += 12;
		}
		if (blackPawnCount[1] >= 1 && blackPawnCount[0] == 0 &&
			blackPawnCount[2] == 0)
		{
			board.Score += 14;
		}
		if (blackPawnCount[2] >= 1 && blackPawnCount[1] == 0 &&
			blackPawnCount[3] == 0)
		{
			board.Score += 16;
		}
		if (blackPawnCount[3] >= 1 && blackPawnCount[2] == 0 &&
			blackPawnCount[4] == 0)
		{
			board.Score += 20;
		}
		if (blackPawnCount[4] >= 1 && blackPawnCount[3] == 0 &&
			blackPawnCount[5] == 0)
		{
			board.Score += 20;
		}
		if (blackPawnCount[5] >= 1 && blackPawnCount[4] == 0 &&
			blackPawnCount[6] == 0)
		{
			board.Score += 16;
		}
		if (blackPawnCount[6] >= 1 && blackPawnCount[5] == 0 &&
			blackPawnCount[7] == 0)
		{
			board.Score += 14;
		}
		if (blackPawnCount[7] >= 1 && blackPawnCount[6] == 0)
		{
			board.Score += 12;
		}

		//White Isolated Pawns
		if (whitePawnCount[0] >= 1 && whitePawnCount[1] == 0)
		{
			board.Score -= 12;
		}
		if (whitePawnCount[1] >= 1 && whitePawnCount[0] == 0 &&
			whitePawnCount[2] == 0)
		{
			board.Score -= 14;
		}
		if (whitePawnCount[2] >= 1 && whitePawnCount[1] == 0 &&
			whitePawnCount[3] == 0)
		{
			board.Score -= 16;
		}
		if (whitePawnCount[3] >= 1 && whitePawnCount[2] == 0 &&
			whitePawnCount[4] == 0)
		{
			board.Score -= 20;
		}
		if (whitePawnCount[4] >= 1 && whitePawnCount[3] == 0 &&
			whitePawnCount[5] == 0)
		{
			board.Score -= 20;
		}
		if (whitePawnCount[5] >= 1 && whitePawnCount[4] == 0 &&
			whitePawnCount[6] == 0)
		{
			board.Score -= 16;
		}
		if (whitePawnCount[6] >= 1 && whitePawnCount[5] == 0 &&
			whitePawnCount[7] == 0)
		{
			board.Score -= 14;
		}
		if (whitePawnCount[7] >= 1 && whitePawnCount[6] == 0)
		{
			board.Score -= 12;
		}

		//Black Passed Pawns
		if (blackPawnCount[0] >= 1 && whitePawnCount[0] == 0)
		{
			board.Score -= blackPawnCount[0];
		}
		if (blackPawnCount[1] >= 1 && whitePawnCount[1] == 0)
		{
			board.Score -= blackPawnCount[1];
		}
		if (blackPawnCount[2] >= 1 && whitePawnCount[2] == 0)
		{
			board.Score -= blackPawnCount[2];
		}
		if (blackPawnCount[3] >= 1 && whitePawnCount[3] == 0)
		{
			board.Score -= blackPawnCount[3];
		}
		if (blackPawnCount[4] >= 1 && whitePawnCount[4] == 0)
		{
			board.Score -= blackPawnCount[4];
		}
		if (blackPawnCount[5] >= 1 && whitePawnCount[5] == 0)
		{
			board.Score -= blackPawnCount[5];
		}
		if (blackPawnCount[6] >= 1 && whitePawnCount[6] == 0)
		{
			board.Score -= blackPawnCount[6];
		}
		if (blackPawnCount[7] >= 1 && whitePawnCount[7] == 0)
		{
			board.Score -= blackPawnCount[7];
		}

		//White Passed Pawns
		if (whitePawnCount[0] >= 1 && blackPawnCount[1] == 0)
		{
			board.Score += whitePawnCount[0];
		}
		if (whitePawnCount[1] >= 1 && blackPawnCount[1] == 0)
		{
			board.Score += whitePawnCount[1];
		}
		if (whitePawnCount[2] >= 1 && blackPawnCount[2] == 0)
		{
			board.Score += whitePawnCount[2];
		}
		if (whitePawnCount[3] >= 1 && blackPawnCount[3] == 0)
		{
			board.Score += whitePawnCount[3];
		}
		if (whitePawnCount[4] >= 1 && blackPawnCount[4] == 0)
		{
			board.Score += whitePawnCount[4];
		}
		if (whitePawnCount[5] >= 1 && blackPawnCount[5] == 0)
		{
			board.Score += whitePawnCount[5];
		}
		if (whitePawnCount[6] >= 1 && blackPawnCount[6] == 0)
		{
			board.Score += whitePawnCount[6];
		}
		if (whitePawnCount[7] >= 1 && blackPawnCount[7] == 0)
		{
			board.Score += whitePawnCount[7];
		}
		*/
		//std::cout << "Score from cuda evaluation is : " << board.Score << std::endl;
		std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
		time_cuda = time_cuda + std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();
		count_positions = count_positions + 1;
		if (min_time > std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count()) {
			min_time = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();
		}
		if (max_time < std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() && std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() < 160000) {
			max_time = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();
		}
		std::cout << "Time difference gpu = " << time_cuda << "[�s], positions counted: " << count_positions << ", min time: " << min_time << ", max time: " << max_time << " board score: " << board.Score << std::endl;
		hipFree(d_squares), hipFree(d_score); 
		//hipFree(d_Kings), hipFree(d_Queens), hipFree(d_Rooks), hipFree(d_Bishops), hipFree(d_Knights), hipFree(d_Pawns);
		//hipFree(d_PawnTable);
	}
}