#include "hip/hip_runtime.h"
#include "test_gpu.cuh"
#include "Search.h"
#include "Board.h"

__device__ int simple_example(int* temp, int n) {
	printf("Testing");
	return n;
}

__global__ void test_kernel(void) {
}


// Only first kernel launch compiles correctly


__global__ void generate_tree(int* board, int& depth, int board_size, Node* root, pointFunctionGenerateValidMoves GenerateValidMovesPointer, pointFunctionCountMoves CountMovesPointer, int* d_result) {

	int index = threadIdx.x;

	printf("Testing kernel\n");
	// from defaut x starts each game, need to specify that everywhere

	// generate all possible moves from given position, where first element of pointer is moves count


	//int* possible_moves = (*GenerateValidMovesPointer)(board, board_size);
	int* possible_moves = new int[5];
	possible_moves[0] = 1;
	const int len = possible_moves[0];

	printf("Generated possible moves\n");

	// allocate space for all root's children and declare child_count

	root->children = new Node * [len];
	root->children_count = len;

	// check which symbol should be drawn next and declare symbol value: 1 = 'x', -1 = 'o'
	int move_count = (*CountMovesPointer)(root->squares, board_size);
	int symbol = 0;
	if (move_count % 2 == 0) {
		symbol = 1;
	}
	else {
		symbol = -1;
	}

	printf("Calculated symbol\n");

	// for each possible position we need to create new child and add it to root->children array

	Node* child = new Node;
	child->squares = new int[board_size * board_size];

	// we need to copy squares, so root squares will not be changed
	//std::copy(root->squares, root->squares + int(board_size * board_size), child->squares);

	child->squares = root->squares;	// VERY TEMP 

	// adding 1 to array index, because [0] element is moves count, not index of squares array
	child->squares[possible_moves[index + 1]] = symbol;
	child->score = 0;
	child->parent = root;
	root->children[index] = child;
	child->children_count = 0;

	delete[] possible_moves;

	// in small board_size values depth is not needed, we can evaluate all positions, but might be usefull in future implementations

	int* left_cells = (*GenerateValidMovesPointer)(root->children[index]->squares, board_size);
	if (left_cells[0] != 0) {

		// increasing value of depth, because we are going deeper into tree
		depth = depth + 1;
		//generate_tree<<<1, root->children_count>>>(root->children[index]->squares, depth, board_size, child, GenerateValidMovesPointer, CountMovesPointer);
	}

	//std::cout << "Try to delete left_cells" << std::endl;
	delete[] left_cells;

	// std::cout << "Try to delete possible_moves" << std::endl;

	// decreasing depth, because we are returning from recursive algorithm, which means we are going up
	depth = depth - 1;
	*d_result = root->children_count;

	printf("Exiting kernel, childcout: %d\n", root->children_count);
}

namespace Test {

	__device__ pointFunctionCountMoves Pointer_d = simple_example;
	pointFunctionCountMoves Pointer_h;

	void test(void)
	{
		test_kernel <<<1, 1>>> ();
		printf("Hello, CUDA is here!");
	}

	void GenerateTree(int* board, int& depth, int board_size, Node* root) {

		printf("Generate tree device function\n");

		pointFunctionGenerateValidMoves host_GenerateValidMovesPointer;
		pointFunctionGenerateValidMoves GenerateValidMovesPointer = Board::GenerateValidMoves;

		pointFunctionCountMoves host_CountMovesPointer;
		pointFunctionCountMoves CountMovesPointer = Board::CountMoves;

		hipMemcpyFromSymbol(&host_GenerateValidMovesPointer, HIP_SYMBOL(GenerateValidMovesPointer), sizeof(pointFunctionGenerateValidMoves));
		hipMemcpyFromSymbol(&host_CountMovesPointer, HIP_SYMBOL(CountMovesPointer), sizeof(pointFunctionCountMoves));

		int* d_board;
		int *result; 
		int *d_result;
		Node* d_root;

		hipMalloc(&d_board, board_size * sizeof(int));
		hipMalloc(&d_root, sizeof(Node));
		hipMalloc(&d_result, sizeof(int));
		hipMalloc(&result, sizeof(int));

		printf("Result before kernel: %d\n", result);

		hipMemcpy(d_board, board, board_size * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_root, root, sizeof(Node), hipMemcpyHostToDevice);

		generate_tree << <1, 1 >> > (d_board, depth, board_size, d_root, host_GenerateValidMovesPointer, host_CountMovesPointer, d_result);

		hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost);

		printf("Exiting device function\n");
		printf("Children count: %d\n", result);
	}
}