#include "hip/hip_runtime.h"
#include <assert.h>
#include "test_gpu.cuh"
#include "Search.h"
#include "Board.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__device__ int simple_example(int* temp, int n) {
	printf("Testing");
	return n;
}

__global__ void test_kernel(void) {
}

__device__ int CountMovesGPU(int* squares, int size) {
	int counter = 0;
	for (int i = 0; i < size * size; ++i) {
		if (squares[i] != 0) {
			++counter;
		}
	}
	return counter;
}

__device__ int* GenerateValidMovesGPU(int* squares, int board_size) {
	int count_moves = CountMovesGPU(squares, board_size);
	int* free_cells = new int[board_size * board_size - count_moves + 1];
	int counter = 0;

	free_cells[0] = 0;
	for (int i = 0; i < board_size * board_size; ++i) {
		if (IsMoveValidGPU(squares, board_size, i)) {
			counter = counter + 1;
			free_cells[counter] = i;
		}
	}
	free_cells[0] = counter;

	return free_cells;
}

__device__ bool IsMoveValidGPU(int* squares, int board_size, int move) {
	if (IsGameFinishedGPU(squares, board_size) == 0) {
		if (move < board_size * board_size) {
			if (squares[move] == 0) {
				return true;
			}
		}
	}
	return false;
}



__device__ int IsGameFinishedGPU(int* squares, int board_size) {
	int test_sum_row = 0;
	
	// look for win in rows
	for (int i = 0; i < board_size; ++i) {
		test_sum_row = 0;
		for (int j = 0; j < board_size; ++j) {
			if (squares[i * board_size + j] == 1 && test_sum_row >= 0) {
				++test_sum_row;
			}
			else if (squares[i * board_size + j] == -1 && test_sum_row <= 0) {
				--test_sum_row;
			}
			else {
				test_sum_row = 0;
			}

			if (test_sum_row == 3 || test_sum_row == -3) {
				return test_sum_row;
			}
		}
	}
	
	int test_sum_col = 0;

	// look for win in columns
	for (int j = 0; j < board_size; ++j) {
		test_sum_col = 0;
		for (int i = 0; i < board_size; ++i) {
			if (squares[i * board_size + j] == 1 && test_sum_col >= 0) {
				++test_sum_col;
			}
			else if (squares[i * board_size + j] == -1 && test_sum_col <= 0) {
				--test_sum_col;
			}
			else {
				test_sum_col = 0;
			}

			if (test_sum_col == 3 || test_sum_col == -3) {
				return test_sum_col;
			}
		}
	}

	int test_sum_diagonal = 0;

	// look for win in diagonals
	// for now only support 3x3 boards and their main diagonals
	for (int i = 0; i < board_size; ++i) {
		if (squares[i * board_size + i] == 1 && test_sum_diagonal >= 0) {
			++test_sum_diagonal;
		}
		else if (squares[i * board_size + i] == -1 && test_sum_diagonal <= 0) {
			--test_sum_diagonal;
		}
		else {
			test_sum_diagonal = 0;
		}

		if (test_sum_diagonal == 3 || test_sum_diagonal == -3) {
			return test_sum_diagonal;
		}
	}

	test_sum_diagonal = 0;

	for (int i = 0; i < board_size; ++i) {
		if (squares[i * board_size + (board_size - 1 - i)] == 1 && test_sum_diagonal >= 0) {
			++test_sum_diagonal;
		}
		else if (squares[i * board_size + (board_size - 1 - i)] == -1 && test_sum_diagonal <= 0) {
			--test_sum_diagonal;
		}
		else {
			test_sum_diagonal = 0;
		}

		if (test_sum_diagonal == 3 || test_sum_diagonal == -3) {
			return test_sum_diagonal;
		}
	}


	int test_draw = 1;
	for (int i = 0; i < board_size * board_size; ++i) {
		test_draw = test_draw * squares[i];
	}

	return test_draw;
}

__device__ pointFunctionGenerateValidMoves GenerateValidMovesPointer = GenerateValidMovesGPU;
__device__ pointFunctionCountMoves CountMovesPointer = CountMovesGPU;

// Only first kernel launch compiles correctly
// There is issue with GenerateValidMovesPointer, propably it needs device word in initialization, propably same problem with CountMovesPointer


__global__ void generate_tree(int* board, int depth, int board_size, Node* root, int* root_squares, Node* root_parent, int root_score, pointFunctionGenerateValidMoves GenerateValidMovesPointer, pointFunctionCountMoves CountMovesPointer, int* d_result) {

	int index = threadIdx.x;

	root->score = root_score;
	root->squares = root_squares;
	root->parent = root_parent;

	printf("Testing kernel, index: %d, depth: %d, pos:\n", index, depth);
	printf(" %d  %d  %d\n", board[0], board[1], board[2]);
	printf(" %d  %d  %d\n", board[3], board[4], board[5]);
	printf(" %d  %d  %d\n", board[6], board[7], board[8]);
	// from defaut x starts each game, need to specify that everywhere

	// generate all possible moves from given position, where first element of pointer is moves count


	int* possible_moves = (*GenerateValidMovesPointer)(board, board_size);
	const int len = possible_moves[0];

	// allocate space for all root's children and declare child_count

	root->children = new Node * [len];
	root->children_count = len;

	printf("Children count: %d, index: %d, depth: %d\n", root->children_count, index, depth);

	// check which symbol should be drawn next and declare symbol value: 1 = 'x', -1 = 'o'
	printf("Before counting moves, index: %d, depth: %d, pos\n", index, depth);
	printf(" %d  %d  %d\n", root->squares[0], root->squares[1], root->squares[2]);
	printf(" %d  %d  %d\n", root->squares[3], root->squares[4], root->squares[5]);
	printf(" %d  %d  %d\n", root->squares[6], root->squares[7], root->squares[8]);
	int move_count = (*CountMovesPointer)(root->squares, board_size);
	int symbol = 0;
	if (move_count % 2 == 0) {
		symbol = 1;
	}
	else {
		symbol = -1;
	}

	//printf("Calculated symbol: %d\n", symbol);

	// for each possible position we need to create new child and add it to root->children array

	Node* child = new Node;
	child->squares = new int[board_size * board_size];

	// we need to copy squares, so root squares will not be changed
	//std::copy(root->squares, root->squares + int(board_size * board_size), child->squares);

	child->squares = root->squares;	// VERY TEMP 

	// adding 1 to array index, because [0] element is moves count, not index of squares array
	child->squares[possible_moves[index + 1]] = symbol;
	child->score = 0;
	child->parent = root;
	root->children[index] = child;
	child->children_count = 0;

	delete[] possible_moves;

	//printf("Created children\n");

	// in small board_size values depth is not needed, we can evaluate all positions, but might be usefull in future implementations

	int** left_cells = new int* [index];

	left_cells[index] = (*GenerateValidMovesPointer)(root->children[index]->squares, board_size);

	printf("Generated next len: %d, index: %d, depth: %d\n", left_cells[index][0], index, depth);
	if (left_cells[index][0] != 0) {
		//printf("In if\n");
		// increasing value of depth, because we are going deeper into tree
		depth = depth + 1;
		generate_tree<<<1, root->children_count>>>(root->children[index]->squares, depth, board_size, root->children[index], root->children[index]->squares, root->children[index]->parent, root->children[index]->score, GenerateValidMovesPointer, CountMovesPointer, d_result);
		hipDeviceSynchronize();
	}
	//printf("After if\n");

	//std::cout << "Try to delete left_cells" << std::endl;
	delete[] left_cells;

	// std::cout << "Try to delete possible_moves" << std::endl;

	// decreasing depth, because we are returning from recursive algorithm, which means we are going up
	depth = depth - 1;
	*d_result = root->children_count;

	printf("Exiting kernel, child_count: %d, index: %d, depth: %d\n", *d_result, index, depth);
}

namespace Test {

	__device__ pointFunctionCountMoves Pointer_d = simple_example;
	pointFunctionCountMoves Pointer_h;

	void test(void)
	{
		test_kernel <<<1, 1>>> ();
		printf("Hello, CUDA is here!");
	}

	void GenerateTree(int* board, int& depth, int board_size, Node* root) {

		printf("Generate tree device function\n");

		pointFunctionGenerateValidMoves host_GenerateValidMovesPointer;
		pointFunctionCountMoves host_CountMovesPointer;

		hipMemcpyFromSymbol(&host_GenerateValidMovesPointer, HIP_SYMBOL(GenerateValidMovesPointer), sizeof(pointFunctionGenerateValidMoves));
		hipMemcpyFromSymbol(&host_CountMovesPointer, HIP_SYMBOL(CountMovesPointer), sizeof(pointFunctionCountMoves));

		int* d_board;
		int *result = (int*)malloc(sizeof(int)); 
		int *d_result;
		int* root_squares;
		Node* root_parent;
		Node* d_root;

		gpuErrchk(hipMalloc(&d_board, board_size * board_size * sizeof(int)));
		gpuErrchk(hipMalloc(&d_result, sizeof(int)));
		gpuErrchk(hipMalloc(&root_squares, board_size * board_size * sizeof(int)));
		gpuErrchk(hipMalloc(&root_parent, sizeof(Node)));
		gpuErrchk(hipMalloc(&d_root, sizeof(Node)));

		printf("Result before kernel: %d\n", result);

		hipMemcpy(d_board, board, board_size * board_size * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_root, root, sizeof(Node), hipMemcpyHostToDevice);
		hipMemcpy(root_squares, root->squares, board_size * board_size * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(root_parent, root->parent, sizeof(Node), hipMemcpyHostToDevice);
		hipMemcpy(d_root, root, sizeof(Node), hipMemcpyHostToDevice);

		generate_tree << <1, 1 >> > (d_board, depth, board_size, d_root, root_squares, root_parent, root->score, host_GenerateValidMovesPointer, host_CountMovesPointer, d_result);
		//gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());

		printf("Exited kernel\n");

		hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost);

		printf("Exiting device function\n");
		printf("Children count: %d\n", *result);
	}
}