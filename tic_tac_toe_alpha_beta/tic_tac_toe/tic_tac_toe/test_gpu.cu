#include "hip/hip_runtime.h"
#include "test_gpu.cuh"
#include "Search.h"
#include "Board.h"

typedef int* (*pointFunctionGenerateValidMoves)(int*, int);
typedef int (*pointFunctionCountMoves)(int*, int);

__device__ int simple_example(int* temp, int n) {
	printf("Testing");
	return n;
}

__global__ void test_kernel(void) {
}

// pointFunctionGenerateValidMoves GenerateValidMovesPointer, pointFunctionCountMoves CountMovesPointer

__global__ void generate_tree(int* board, int& depth, int board_size, Node* root, pointFunctionCountMoves Pointer) {
	
	int index = threadIdx.x;
	int *temp = new int;
	int n = 0;

	n = (*Pointer)(temp, n);

	//printf("Testing generating gametree\n");
	// from defaut x starts each game, need to specify that everywhere

	// generate all possible moves from given position, where first element of pointer is moves count
	
	/*
	int* possible_moves = (*GenerateValidMovesPointer)(board, board_size);
	const int len = possible_moves[0];

	// allocate space for all root's children and declare child_count

	root->children = new Node * [len];
	root->children_count = len;

	// check which symbol should be drawn next and declare symbol value: 1 = 'x', -1 = 'o'
	int move_count = (*CountMovesPointer)(root->squares, board_size);
	int symbol = 0;
	if (move_count % 2 == 0) {
		symbol = 1;
	}
	else {
		symbol = -1;
	}

	// for each possible position we need to create new child and add it to root->children array

	Node* child = new Node;
	child->squares = new int[board_size * board_size];

	// we need to copy squares, so root squares will not be changed
	//std::copy(root->squares, root->squares + int(board_size * board_size), child->squares);

	child->squares = root->squares;	// VERY TEMP 

	// adding 1 to array index, because [0] element is moves count, not index of squares array
	child->squares[possible_moves[index + 1]] = symbol;
	child->score = 0;
	child->parent = root;
	root->children[index] = child;
	child->children_count = 0;

	delete[] possible_moves;

	// in small board_size values depth is not needed, we can evaluate all positions, but might be usefull in future implementations

	int* left_cells = (*GenerateValidMovesPointer)(root->children[index]->squares, board_size);
	if (left_cells[0] != 0) {

		// increasing value of depth, because we are going deeper into tree
		depth = depth + 1;
		//generate_tree<<<1, root->children_count>>>(root->children[index]->squares, depth, board_size, child, GenerateValidMovesPointer, CountMovesPointer);
	}

	//std::cout << "Try to delete left_cells" << std::endl;
	delete[] left_cells;

	// std::cout << "Try to delete possible_moves" << std::endl;

	*/

	// decreasing depth, because we are returning from recursive algorithm, which means we are going up
	depth = depth - 1;
}

namespace Test {

	//__device__ pointFunctionGenerateValidMoves GenerateValidMovesPointer = Board::GenerateValidMoves;
	//pointFunctionGenerateValidMoves host_GenerateValidMovesPointer;

	//__device__ pointFunctionCountMoves CountMovesPointer = Board::CountMoves;
	//pointFunctionCountMoves host_CountMovesPointer;

	__device__ pointFunctionCountMoves Pointer_d = simple_example;
	pointFunctionCountMoves Pointer_h;

	void test(void)
	{
		test_kernel <<<1, 1>>> ();
		printf("Hello, CUDA is here!");
	}

	void GenerateTree(int* board, int& depth, int board_size, Node* root) {

		//hipMemcpyFromSymbol(&host_GenerateValidMovesPointer, HIP_SYMBOL(GenerateValidMovesPointer), sizeof(pointFunctionGenerateValidMoves));
		//hipMemcpyFromSymbol(&host_CountMovesPointer, HIP_SYMBOL(CountMovesPointer), sizeof(pointFunctionCountMoves));

		hipMemcpyFromSymbol(&Pointer_h, HIP_SYMBOL(Pointer_d), sizeof(pointFunctionCountMoves));


		//generate_tree << <1, 1 >> > (board, depth, board_size, root, host_GenerateValidMovesPointer, host_CountMovesPointer);

		generate_tree << <1, 1 >> > (board, depth, board_size, root, Pointer_h);
	}
}