#include "hip/hip_runtime.h"
#include <assert.h>
#include "test_gpu.cuh"
#include "Search.h"
#include "Board.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__device__ int simple_example(int* temp, int n) {
	printf("Testing");
	return n;
}

__global__ void test_kernel(void) {
}

__device__ int CountMovesGPU(int* squares, int size) {
	int counter = 0;
	//printf("CountMovesGPU\n %d  %d  %d\n %d  %d  %d\n %d  %d  %d\n", squares[0], squares[1], squares[2], squares[3], squares[4], squares[5], squares[6], squares[7], squares[8]);
	//printf(" %d  %d  %d\n", squares[3], squares[4], squares[5]);
	//printf(" %d  %d  %d\n", squares[6], squares[7], squares[8]);
	for (int i = 0; i < size * size; ++i) {
		if (squares[i] != 0) {
			++counter;
		}
	}
	return counter;
}

__device__ int* GenerateValidMovesGPU(int* squares, int board_size) {
	int count_moves = CountMovesGPU(squares, board_size);
	int* free_cells = new int[board_size * board_size - count_moves + 1];
	int counter = 0;
	//printf("Counted moves %d\n", count_moves);

	free_cells[0] = 0;
	for (int i = 0; i < board_size * board_size; ++i) {
		if (IsMoveValidGPU(squares, board_size, i)) {
			counter = counter + 1;
			free_cells[counter] = i;
		}
	}
	free_cells[0] = counter;

	return free_cells;
}

__device__ bool IsMoveValidGPU(int* squares, int board_size, int move) {
	if (IsGameFinishedGPU(squares, board_size) == 0) {
		if (move < board_size * board_size) {
			if (squares[move] == 0) {
				return true;
			}
		}
	}
	return false;
}

__device__ int IsGameFinishedGPU(int* squares, int board_size) {
	int test_sum_row = 0;
	
	// look for win in rows
	for (int i = 0; i < board_size; ++i) {
		test_sum_row = 0;
		for (int j = 0; j < board_size; ++j) {
			if (squares[i * board_size + j] == 1 && test_sum_row >= 0) {
				++test_sum_row;
			}
			else if (squares[i * board_size + j] == -1 && test_sum_row <= 0) {
				--test_sum_row;
			}
			else {
				test_sum_row = 0;
			}

			if (test_sum_row == 3 || test_sum_row == -3) {
				return test_sum_row;
			}
		}
	}
	
	int test_sum_col = 0;

	// look for win in columns
	for (int j = 0; j < board_size; ++j) {
		test_sum_col = 0;
		for (int i = 0; i < board_size; ++i) {
			if (squares[i * board_size + j] == 1 && test_sum_col >= 0) {
				++test_sum_col;
			}
			else if (squares[i * board_size + j] == -1 && test_sum_col <= 0) {
				--test_sum_col;
			}
			else {
				test_sum_col = 0;
			}

			if (test_sum_col == 3 || test_sum_col == -3) {
				return test_sum_col;
			}
		}
	}

	int test_sum_diagonal = 0;

	// look for win in diagonals
	// for now only support 3x3 boards and their main diagonals
	for (int i = 0; i < board_size; ++i) {
		if (squares[i * board_size + i] == 1 && test_sum_diagonal >= 0) {
			++test_sum_diagonal;
		}
		else if (squares[i * board_size + i] == -1 && test_sum_diagonal <= 0) {
			--test_sum_diagonal;
		}
		else {
			test_sum_diagonal = 0;
		}

		if (test_sum_diagonal == 3 || test_sum_diagonal == -3) {
			return test_sum_diagonal;
		}
	}

	test_sum_diagonal = 0;

	for (int i = 0; i < board_size; ++i) {
		if (squares[i * board_size + (board_size - 1 - i)] == 1 && test_sum_diagonal >= 0) {
			++test_sum_diagonal;
		}
		else if (squares[i * board_size + (board_size - 1 - i)] == -1 && test_sum_diagonal <= 0) {
			--test_sum_diagonal;
		}
		else {
			test_sum_diagonal = 0;
		}

		if (test_sum_diagonal == 3 || test_sum_diagonal == -3) {
			return test_sum_diagonal;
		}
	}


	int test_draw = 1;
	for (int i = 0; i < board_size * board_size; ++i) {
		test_draw = test_draw * squares[i];
	}

	return test_draw;
}


__device__ void GenerateTreeParrarel(int* board, int& depth, int board_size, Node* root) {

	// from defaut x starts each game, need to specify that everywhere
	//printf("Depth: %d\n", depth);
	// generate all possible moves from given position, where first element of pointer is moves count
	int* possible_moves = GenerateValidMovesGPU(board, board_size);
	const int len = possible_moves[0];

	// allocate space for all root's children and declare child_count

	root->children = new Node * [len];
	root->children_count = len;

	// check which symbol should be drawn next and declare symbol value: 1 = 'x', -1 = 'o'
	int move_count = CountMovesGPU(root->squares, board_size);
	int symbol = 0;
	if (move_count % 2 == 0) {
		symbol = 1;
	}
	else {
		symbol = -1;
	}

	// for each possible position we need to create new child and add it to root->children array
	for (int i = 0; i < len; ++i) {

		Node* child = new Node;
		child->squares = new int[board_size * board_size];

		// we need to copy squares, so root squares will not be changed
		//std::copy(root->squares, root->squares + int(board_size * board_size), child->squares);
		memcpy(child->squares, root->squares, board_size * board_size * sizeof(int));

		// adding 1 to array index, because [0] element is moves count, not index of squares array
		child->squares[possible_moves[i + 1]] = symbol;
		child->score = 0;
		child->parent = root;
		root->children[i] = child;
		child->children_count = 0;

		// in small board_size values depth is not needed, we can evaluate all positions, but might be usefull in future implementations

		int* left_cells = GenerateValidMovesGPU(child->squares, board_size);
		if (left_cells[0] != 0) {

			// increasing value of depth, because we are going deeper into tree
			depth = depth + 1;
			GenerateTreeParrarel(child->squares, depth, board_size, child);
		}

		//std::cout << "Try to delete left_cells" << std::endl;
		delete[] left_cells;
	}

	//std::cout << "Try to delete possible_moves" << std::endl;
	delete[] possible_moves;

	// decreasing depth, because we are returning from recursive algorithm, which means we are going up
	depth = depth - 1;
}

__device__ pointFunctionGenerateValidMoves GenerateValidMovesPointer = GenerateValidMovesGPU;
__device__ pointFunctionCountMoves CountMovesPointer = CountMovesGPU;
__device__ pointFunctionGenerateTreeParrarel GenerateTreeParrarelPointer = GenerateTreeParrarel;

// Only first kernel launch compiles correctly
// There is issue with GenerateValidMovesPointer, propably it needs device word in initialization, propably same problem with CountMovesPointer


__global__ void generate_tree(int* board, int depth, int board_size, Node* root, int* root_squares, Node* root_parent, int root_score, pointFunctionGenerateValidMoves GenerateValidMovesPointer, pointFunctionCountMoves CountMovesPointer, int* d_result) {

	int index = threadIdx.x;

	root->score = root_score;
	root->squares = root_squares;
	root->parent = root_parent;

	//printf("Testing kernel, depth: %d, pos:\n", depth);
	//printf(" %d  %d  %d\n %d  %d  %d\n %d  %d  %d\n", root->squares[0], root->squares[1], root->squares[2], root->squares[3], root->squares[4], root->squares[5], root->squares[6], root->squares[7], root->squares[8]);
		//printf(" %d  %d  %d\n", root->squares[3], root->squares[4], root->squares[5]);
		//printf(" %d  %d  %d\n", root->squares[6], root->squares[7], root->squares[8]);

	// from defaut x starts each game, need to specify that everywhere

	// generate all possible moves from given position, where first element of pointer is moves count


	int* possible_moves = (*GenerateValidMovesPointer)(board, board_size);
	const int len = possible_moves[0];

	// allocate space for all root's children and declare child_count

	root->children = new Node * [len];
	root->children_count = len;

	//printf("Children count: %d, index: %d, depth: %d\n", root->children_count, index, depth);

	// check which symbol should be drawn next and declare symbol value: 1 = 'x', -1 = 'o'
	//printf("Before counting moves, index: %d, depth: %d, pos\n", index, depth);
	//printf(" %d  %d  %d\n", root->squares[0], root->squares[1], root->squares[2]);
	//printf(" %d  %d  %d\n", root->squares[3], root->squares[4], root->squares[5]);
	//printf(" %d  %d  %d\n", root->squares[6], root->squares[7], root->squares[8]);
	int move_count = (*CountMovesPointer)(root->squares, board_size);
	int symbol = 0;
	if (move_count % 2 == 0) {
		symbol = 1;
	}
	else {
		symbol = -1;
	}

	//printf("Calculated symbol: %d\n", symbol);

	// for each possible position we need to create new child and add it to root->children array

	Node* child = new Node;
	child->squares = new int[board_size * board_size];

	// we need to copy squares, so root squares will not be changed
	//std::copy(root->squares, root->squares + int(board_size * board_size), child->squares);


	// GOTTA SOMEHOW COPY POSITION
	//child->squares = root->squares;	// VERY TEMP 

	memcpy(child->squares, root->squares, board_size*board_size*sizeof(int));

	/*
	printf("After assigning pos to children[index], index: %d, depth: %d, pos\n", index, depth);
	printf(" %d  %d  %d\n", child->squares[0], child->squares[1], child->squares[2]);
	printf(" %d  %d  %d\n", child->squares[3], child->squares[4], child->squares[5]);
	printf(" %d  %d  %d\n", child->squares[6], child->squares[7], child->squares[8]);

	printf("Addres of child: %d\n", child->squares);
	*/

	// adding 1 to array index, because [0] element is moves count, not index of squares array
	child->squares[possible_moves[index + 1]] = symbol;
	/*
	printf("After adding move to children[index], index: %d, depth: %d, pos\n", index, depth);
	printf(" %d  %d  %d\n", child->squares[0], child->squares[1], child->squares[2]);
	printf(" %d  %d  %d\n", child->squares[3], child->squares[4], child->squares[5]);
	printf(" %d  %d  %d\n", child->squares[6], child->squares[7], child->squares[8]);
	*/
	child->score = 0;
	child->parent = root;
	root->children[index] = child;
	child->children_count = 0;

	delete[] possible_moves;

	//printf("Created children\n");

	// in small board_size values depth is not needed, we can evaluate all positions, but might be usefull in future implementations

	//int** left_cells = new int* [index];

	//printf("Generating left_cells\n");

	int* left_cells = (*GenerateValidMovesPointer)(root->children[index]->squares, board_size);

	//printf("Generated next len: %d, index: %d, depth: %d\n", left_cells[0], index, depth);


	//printf("Index %d, depth %d, %d %d %d %d %d %d %d %d\n", index, depth, left_cells[1], left_cells[2], left_cells[3], left_cells[4], left_cells[5], left_cells[6], left_cells[7], left_cells[8]);


	if (left_cells[0] != 0) {
	//if (depth<6){
		//printf("In if\n");
		// increasing value of depth, because we are going deeper into tree
		depth = depth + 1;
		generate_tree<<<1, left_cells[0]>>>(root->children[index]->squares, depth, board_size, root->children[index], root->children[index]->squares, root->children[index]->parent, root->children[index]->score, GenerateValidMovesPointer, CountMovesPointer, d_result);
		//printf("Exiting depth %d\n", depth);
		hipDeviceSynchronize();
		depth = depth - 1;
	}
	//printf("After if\n");

	//std::cout << "Try to delete left_cells" << std::endl;
	delete[] left_cells;

	// std::cout << "Try to delete possible_moves" << std::endl;

	// decreasing depth, because we are returning from recursive algorithm, which means we are going up
	*d_result = root->children_count;

	//printf("Exiting kernel, child_count: %d, index: %d, depth: %d\n", *d_result, index, depth);
	if (depth == 1 && index == 0) {
		printf("root->children[0]->children[0]->children_count %d:\n", root->children[0]->children[0]->children_count);
	}
}

__global__ void generate_tree_parrarel(int* board, int depth, int board_size, Node* root, int* root_squares, Node* root_parent, int root_score, pointFunctionGenerateValidMoves GenerateValidMovesPointer, pointFunctionCountMoves CountMovesPointer, pointFunctionGenerateTreeParrarel GenerateTreeParrarelPointer, int* d_result) {
	root->score = root_score;
	root->squares = root_squares;
	root->parent = root_parent;
	printf("Running recursive func\n");
	(*GenerateTreeParrarel)(board, depth, board_size, root);
	printf("End recursive func\n");
}

namespace Test {

	__device__ pointFunctionCountMoves Pointer_d = simple_example;
	pointFunctionCountMoves Pointer_h;

	void test(void)
	{
		test_kernel <<<1, 1>>> ();
		printf("Hello, CUDA is here!");
	}

	void GenerateTree(int* board, int& depth, int board_size, Node* root) {

		hipDeviceSetLimit(hipLimitMallocHeapSize, 100000000);
		printf("Generate tree device function\n");

		pointFunctionGenerateValidMoves host_GenerateValidMovesPointer;
		pointFunctionCountMoves host_CountMovesPointer;
		pointFunctionGenerateTreeParrarel host_GenerateTreeParrarelPointer;

		hipMemcpyFromSymbol(&host_GenerateValidMovesPointer, HIP_SYMBOL(GenerateValidMovesPointer), sizeof(pointFunctionGenerateValidMoves));
		hipMemcpyFromSymbol(&host_CountMovesPointer, HIP_SYMBOL(CountMovesPointer), sizeof(pointFunctionCountMoves));
		hipMemcpyFromSymbol(&host_GenerateTreeParrarelPointer, HIP_SYMBOL(GenerateTreeParrarelPointer), sizeof(pointFunctionGenerateTreeParrarel));

		int* d_board;
		int *result = (int*)malloc(sizeof(int)); 
		int *d_result;
		int* root_squares;
		Node* root_parent;
		Node* d_root;

		gpuErrchk(hipMalloc(&d_board, board_size * board_size * sizeof(int)));
		gpuErrchk(hipMalloc(&d_result, sizeof(int)));
		gpuErrchk(hipMalloc(&root_squares, board_size * board_size * sizeof(int)));
		gpuErrchk(hipMalloc(&root_parent, sizeof(Node)));
		gpuErrchk(hipMalloc(&d_root, sizeof(Node)));

		hipMemcpy(d_board, board, board_size * board_size * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_root, root, sizeof(Node), hipMemcpyHostToDevice);
		hipMemcpy(root_squares, root->squares, board_size * board_size * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(root_parent, root->parent, sizeof(Node), hipMemcpyHostToDevice);
		hipMemcpy(d_root, root, sizeof(Node), hipMemcpyHostToDevice);

		printf("Testowanie generowania na karcie nie w kernelu:\n");

		generate_tree_parrarel << <1, 1 >> > (d_board, depth, board_size, d_root, root_squares, root_parent, root->score, host_GenerateValidMovesPointer, host_CountMovesPointer, host_GenerateTreeParrarelPointer, d_result);
		hipDeviceSynchronize();

		printf("Wygenerowano cale drzewo, teraz w kernelu testy zaczac czas:\n");

		generate_tree << <1, 8 >> > (d_board, depth, board_size, d_root, root_squares, root_parent, root->score, host_GenerateValidMovesPointer, host_CountMovesPointer, d_result);
		//gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());

		printf("Exited kernel\n");

		hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost);

		printf("Exiting device function\n");
		printf("Children count: %d\n", *result);
	}
}